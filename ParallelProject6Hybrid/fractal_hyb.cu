/*
Fractal code for CS 4380 / CS 5351

Copyright (c) 2019 Texas State University. All rights reserved.

Redistribution in source or binary form, with or without modification,
is *not* permitted. Use in source and binary forms, with or without
modification, is only permitted for academic use in CS 4380 or CS 5351
at Texas State University.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher
*/

#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>

static const int ThreadsPerBlock = 512;

static __global__ void fractal(const int width, const int start_frame, const int gpu_frames, unsigned char* const pic)
{
  // todo: use the GPU to compute the requested frames (base the code on the previous project)
}

unsigned char* GPU_Init(const int gpu_frames, const int width)
{
  unsigned char* d_pic;
  if (hipSuccess != hipMalloc((void **)&d_pic, gpu_frames * width * width * sizeof(unsigned char))) {fprintf(stderr, "ERROR: could not allocate memory\n"); exit(-1);}
  return d_pic;
}

void GPU_Exec(const int start_frame, const int gpu_frames, const int width, unsigned char* d_pic)
{
  // todo: launch the kernel with ThreadsPerBlock and the appropriate number of blocks (do not wait for the kernel to finish)
}

void GPU_Fini(const int gpu_frames, const int width, unsigned char* pic, unsigned char* d_pic)
{
  // todo: copy the result from the device to the host and free the device memory
}

